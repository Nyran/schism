#include "hip/hip_runtime.h"

// Copyright (c) 2012 Christopher Lux <christopherlux@gmail.com>
// Distributed under the Modified BSD License, see license.txt.

#include "volume_ray_cast.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cutil/cutil_math.h>

#include <renderer/volume_uniform_data.h>

#define SCM_LDATA_CUDA_VIS_PROFILE_CLOCK 0 
#define SCM_LDATA_CUDA_VIS_ITER_COUNT    0
#define SCM_LDATA_CUDA_VIS_DEBUG         0

#define SCM_LDATA_CUDA_VIS_SS_COUNT      4 // supported modes: 4, 8

// cuda globals
surface<void, cudaSurfaceType2D> out_image;

texture<unsigned char, hipTextureType3D, hipReadModeNormalizedFloat> volume_texture;
texture<uchar4,        hipTextureType1D, hipReadModeNormalizedFloat> colormap_texture;
//texture<float,  hipTextureType3D, hipReadModeElementType> volume_texture;
//texture<float4, hipTextureType1D, hipReadModeElementType> colormap_texture;

__device__ __constant__ volume_uniform_data uniform_data;

// helpers
inline __device__ float4 min(float4 a, float4 b) { return make_float4(min(a.x, b.x), min(a.y, b.y), min(a.z, b.z), min(a.w, b.w)); }
inline __device__ float4 max(float4 a, float4 b) { return make_float4(max(a.x, b.x), max(a.y, b.y), max(a.z, b.z), max(a.w, b.w)); }
inline __device__ float3 min(float3 a, float3 b) { return make_float3(min(a.x, b.x), min(a.y, b.y), min(a.z, b.z)); }
inline __device__ float3 max(float3 a, float3 b) { return make_float3(max(a.x, b.x), max(a.y, b.y), max(a.z, b.z)); }

inline
__device__
float4
mul_matrix4(const float4x4 m, const float4 v)
{
    return make_float4(dot(v, m.rows[0]),  //v), //
                       dot(v, m.rows[1]),  //v), //
                       dot(v, m.rows[2]),  //v), //
                       dot(v, m.rows[3])); //v));//
}                                          
                                           
struct ray                                 
{
    float3  origin;
    float3  direction;
    float3  direction_rec;
}; // struct ray

inline
__device__
void
make_ray(struct ray*const r,
         const float2 spos,
         const int2   ssize)
{
    float4 spos_nrm = make_float4((spos.x / (float)ssize.x) * 2.0f - 1.0f,
                                  (spos.y / (float)ssize.y) * 2.0f - 1.0f,
                                  -1.0f,
                                   1.0f);
    //float4 spos_os  = mul_matrix4_ptr(&(vdata->_mvp_matrix_inverse), &spos_nrm);
    float4 spos_os  = mul_matrix4(uniform_data._mvp_matrix_inverse, spos_nrm);
    spos_os /= spos_os.w;

    r->origin        = make_float3(uniform_data._os_camera_position);//.xyz;
    r->direction     = normalize(make_float3(spos_os) - r->origin);//vdata->_mvp_matrix_inverse.s012;//spos_os.xyz;//
    r->direction_rec = 1.0f / r->direction;
}

bool
__device__
ray_box_intersection(const struct ray*const r,
                     float3   bbmin,
                     float3   bbmax,
                     float*   tmin,
                     float*   tmax)
{
#if 1
    // compute intersection of ray with all six bbox planes
    float3 tbot = r->direction_rec * (bbmin - r->origin);
    float3 ttop = r->direction_rec * (bbmax - r->origin);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin_a = min(ttop, tbot);
    float3 tmax_a = max(ttop, tbot);

    // find the largest tmin and the smallest tmax
    *tmin = max(max(tmin_a.x, tmin_a.y), max(tmin_a.x, tmin_a.z));
    *tmax = min(min(tmax_a.x, tmax_a.y), min(tmax_a.x, tmax_a.z));
#else
    float l1 = (bbmin.x - r->origin.x) * r->direction_rec.x;
    float l2 = (bbmax.x - r->origin.x) * r->direction_rec.x;
    *tmin = min(l1,l2);
    *tmax = max(l1,l2);

    l1   = (bbmin.y - r->origin.y) * r->direction_rec.y;
    l2   = (bbmax.y - r->origin.y) * r->direction_rec.y;
    *tmin = max(min(l1,l2), *tmin);
    *tmax = min(max(l1,l2), *tmax);
        
    l1   = (bbmin.z - r->origin.z) * r->direction_rec.z;
    l2   = (bbmax.z - r->origin.z) * r->direction_rec.z;
    *tmin = max(min(l1,l2), *tmin);
    *tmax = min(max(l1,l2), *tmax);

    //return ((lmax > 0.f) & (lmax >= lmin));
    //return ((lmax > 0.f) & (lmax > lmin));
#endif
    return ((*tmin > 0.0) && (*tmax > *tmin));
}

inline
__device__
float
length_sqr(const float3 a, const float3 b)
{
    float3 d = b - a;
    //return mad(d.x, d.x, mad(d.y, d.y, d.z * d.z));
    return dot(d, d);
}


extern "C"
void
__global__
main_vrc(unsigned out_image_w, unsigned out_image_h, bool use_ss)
{
#if SCM_LDATA_CUDA_VIS_PROFILE_CLOCK == 1
    clock_t thread_start;
    clock_t thread_stop;
#endif // SCM_LDATA_CUDA_VIS_PROFILE_CLOCK == 1

    int2 osize = make_int2(out_image_w, out_image_h);
    int2 opos  = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                           blockIdx.y * blockDim.y + threadIdx.y);

    const int    ss_count      = use_ss ? SCM_LDATA_CUDA_VIS_SS_COUNT : 1;
#if SCM_LDATA_CUDA_VIS_SS_COUNT == 4
    // regular grid
    //const float2 ss_pixel_offsets[4] = {{0.25f, 0.25f},
    //                                    {0.75f, 0.25f},
    //                                    {0.25f, 0.75f},
    //                                    {0.75f, 0.75f}};
    // rotated grid grid
    const float  ss_grid_res = 0.125f;
    const float2 ss_pixel_offsets[4] = {{ss_grid_res * 5.0f, ss_grid_res * 1.0f},
                                        {ss_grid_res * 7.0f, ss_grid_res * 5.0f},
                                        {ss_grid_res * 3.0f, ss_grid_res * 7.0f},
                                        {ss_grid_res * 1.0f, ss_grid_res * 3.0f}};
    const float ss_sample_offsets[4] = {0.00f,
                                        0.25f,
                                        0.50f,
                                        0.75f};
    struct ray ss_rays[4];
#elif SCM_LDATA_CUDA_VIS_SS_COUNT == 8
    // NV pattern
    const float2 ss_pixel_offsets[8] = {{0.630f, 0.206f},
                                        {0.667f, 0.079f},
                                        {0.413f, 0.333f},
                                        {0.794f, 0.460f},
                                        {0.032f, 0.587f},
                                        {0.531f, 0.714f},
                                        {0.286f, 0.841f},
                                        {0.921f, 0.968f}};
    const float ss_sample_offsets[8] = {0.000f,
                                        0.125f,
                                        0.250f,
                                        0.375f,
                                        0.500f,
                                        0.625f,
                                        0.750f,
                                        0.875f};
    //struct ray ss_rays[8];
#endif

    if (opos.x < osize.x && opos.y < osize.y) {

#if SCM_LDATA_CUDA_VIS_PROFILE_CLOCK == 1
        thread_start = clock();
#endif // SCM_LDATA_CUDA_VIS_PROFILE_CLOCK == 1
        float4 out_color = make_float4(0.0);;

#if 1
        struct ray_state {
            ray         _ray;       // the sub-pixel ray
            float       _t;
            float4      _cdst;      // destination color 
            float2      _trange;    // the t min/max range of the ray

        };
        ray_state ray_states[SCM_LDATA_CUDA_VIS_SS_COUNT];
        bool any_ray_running = false;

        // setup rays
        if (use_ss) {
            for (int i = 0; i < ss_count; ++i) {
                const float2 opos_pc = ss_pixel_offsets[i] + make_float2(opos.x, opos.y);
                make_ray(&(ray_states[i]._ray), opos_pc, osize);

                ray_states[i]._cdst = make_float4(0.0f);

                if (ray_box_intersection(&(ray_states[i]._ray),
                                         make_float3(0.0),
                                         make_float3(uniform_data._volume_extends),
                                         &(ray_states[i]._trange.x),
                                         &(ray_states[i]._trange.y)))
                {
                    ray_states[i]._t =   ray_states[i]._trange.x
                                       + ss_sample_offsets[i] * uniform_data._sampling_distance.x;
                    any_ray_running = true;
                }
                else {
                    ray_states[i]._t = ray_states[i]._trange.y;
                    ray_states[i]._cdst = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
                }
            }
        }
        else {
            const float2 opos_pc = make_float2(0.5f + opos.x, 0.5f + opos.y);
            make_ray(&(ray_states[0]._ray), opos_pc, osize);

            ray_states[0]._cdst = make_float4(0.0f);

            if (ray_box_intersection(&(ray_states[0]._ray),
                                     make_float3(0.0),
                                     make_float3(uniform_data._volume_extends),
                                     &(ray_states[0]._trange.x),
                                     &(ray_states[0]._trange.y)))
            {
                ray_states[0]._t = ray_states[0]._trange.x;
                any_ray_running = true;
            }
            else {
                ray_states[0]._t = ray_states[0]._trange.y;
            }
        }

        const float3 obj_to_tex  = make_float3(uniform_data._scale_obj_to_tex);
        const float  op_corr     = uniform_data._sampling_distance.y;
        const float  s_dist      = uniform_data._sampling_distance.x;

        while (any_ray_running) {
            any_ray_running = false;
            for (int s = 0; s < ss_count; ++s) {
                ray_state& r = ray_states[s];

                if (   r._t < r._trange.y
                    && r._cdst.w < 0.99f)
                {
                    any_ray_running = true;
                    const float3 spos      = r._ray.origin + r._t * r._ray.direction;
                    const float3 vtexcoord = obj_to_tex * spos;

                    const float  s   = tex3D(volume_texture, vtexcoord.x, vtexcoord.y, vtexcoord.z);
                    float4 src = tex1D(colormap_texture, s);

                    // advance ray
                    r._t += s_dist;

                    // opacity correction
                    src.w = 1.0f - pow(1.0f - src.w, op_corr);

                    // compositing
                    float omda_sa = (1.0 - r._cdst.w) * src.w;
                    r._cdst.x += omda_sa * src.x;
                    r._cdst.y += omda_sa * src.y;
                    r._cdst.z += omda_sa * src.z;
                    r._cdst.w += omda_sa;
                }
            }
        }

        for (int s = 0; s < ss_count; ++s) {
            out_color += ray_states[s]._cdst;
        }

#else
        // setup rays
        if (use_ss) {
            for (int i = 0; i < ss_count; ++i) {
                const float2 opos_pc = ss_pixel_offsets[i] + make_float2(opos.x, opos.y);
                make_ray(&(ss_rays[i]), opos_pc, osize);
            }
        }
        else {
            const float2 opos_pc = make_float2(0.5f + opos.x, 0.5f + opos.y);
            make_ray(&(ss_rays[0]), opos_pc, osize);
        }

        for (int i = 0; i < ss_count; ++i) {
            float tmin = 0.0;
            float tmax = 0.0;
        
            if (ray_box_intersection(&(ss_rays[i]), make_float3(0.0), make_float3(uniform_data._volume_extends), &tmin, &tmax)) {

                const struct ray& cur_ray = ss_rays[i];
                float3 cam_pos   = make_float3(uniform_data._os_camera_position);
                float3 ray_entry = tmin * cur_ray.direction + cur_ray.origin;
                float3 ray_exit  = tmax * cur_ray.direction + cur_ray.origin;

                float3 ray_increment = cur_ray.direction * uniform_data._sampling_distance.x;
                float3 sampling_pos  = ray_entry + ray_increment; // test, increment just to be sure we are in the volume
                if (use_ss) {
                    sampling_pos += ray_increment * ss_sample_offsets[i];
                }
                float3 to_tex        = make_float3(uniform_data._scale_obj_to_tex);

                float smpl_sqr_dist  = length_sqr(cam_pos, sampling_pos);
                float exit_sqr_dist  = length_sqr(cam_pos, ray_exit);

                float4 dst = make_float4(0.0f);
                float  opc = uniform_data._sampling_distance.y;
                int    loop_count = 0;

                //out_color = make_float4(ray_exit, 1.0);
                while ((exit_sqr_dist - smpl_sqr_dist) > 0.0f && dst.w < 0.99f) {
                    ++loop_count;
                    float3 tc_vol = sampling_pos * to_tex;

                    float  s   = tex3D(volume_texture, tc_vol.x, tc_vol.y, tc_vol.z);// texture(volume_raw, sampling_pos * volume_data.scale_obj_to_tex.xyz).r;
                    float4 src = tex1D(colormap_texture, s);
                    //float4 src    = read_imagef(volume_image, vol_smpl, tc_vol).xxxx;//(float4)(s);//texture(color_map, s);

                    //float4 src = (float4)(s, s, s, 0.1);

                    // increment ray
                    sampling_pos  += ray_increment;
                    smpl_sqr_dist  = length_sqr(cam_pos, sampling_pos);

                    //float3 d = cam_pos - sampling_pos;
                    //smpl_sqr_dist  = dot(d, d);

                    //inside_volume  = inside_volume_bounds(sampling_pos) && (dst.a < 0.99);

                    // opacity correction
                    src.w = 1.0f - pow(1.0f - src.w, opc);

                    // compositing
                    float omda_sa = (1.0 - dst.w) * src.w;
                    dst.x += omda_sa * src.x;
                    dst.y += omda_sa * src.y;
                    dst.z += omda_sa * src.z;
                    dst.w   += omda_sa;
                }
                out_color += dst;
            }
            //else {
            //    out_color += make_float4(1.0f, 0.0f, 0.0f, 1.0f);
            //}
        }
#endif
#if SCM_LDATA_CUDA_VIS_PROFILE_CLOCK == 1
        thread_stop = clock();
        out_color = tex1D(colormap_texture, (float)(thread_stop - thread_start) / 3000000.0f);
#else
        out_color /= float(ss_count);
#endif // SCM_LDATA_CUDA_VIS_PROFILE_CLOCK == 1

        uchar4 out_col_data;
        out_col_data.x = (unsigned char)(out_color.x * 255.0f);
        out_col_data.y = (unsigned char)(out_color.y * 255.0f);
        out_col_data.z = (unsigned char)(out_color.z * 255.0f);
        out_col_data.w = (unsigned char)(out_color.w * 255.0f);

        surf2Dwrite(out_col_data, out_image, opos.x * sizeof(uchar4), opos.y);
    }
}

extern "C"
void
startup_ray_cast_kernel(unsigned out_image_w, unsigned out_image_h,
                        hipGraphicsResource_t                   output_image_res,
                        hipGraphicsResource_t                   volume_image_res,
                        hipGraphicsResource_t                   cmap_image_res,
                        bool                                     use_supersampling,
                        hipStream_t                             hip_stream)
{
    hipError_t   cu_err = hipSuccess;

    // output image
    hipArray*             cu_oi_array = 0;
    cu_err = hipGraphicsSubResourceGetMappedArray(&cu_oi_array, output_image_res, 0, 0);
    cu_err = cudaBindSurfaceToArray(out_image, cu_oi_array);

    // volume texture
    volume_texture.addressMode[0] = hipAddressModeClamp;
    volume_texture.addressMode[1] = hipAddressModeClamp;
    volume_texture.addressMode[2] = hipAddressModeClamp;
    volume_texture.filterMode     = hipFilterModeLinear;
    volume_texture.normalized     = true;
    hipArray* cu_vi_array = 0;
    cu_err = hipGraphicsSubResourceGetMappedArray(&cu_vi_array, volume_image_res, 0, 0);
    cu_err = hipBindTextureToArray(volume_texture, cu_vi_array);

    // color map texture
    colormap_texture.addressMode[0] = hipAddressModeClamp;
    colormap_texture.filterMode     = hipFilterModeLinear;
    colormap_texture.normalized     = true;
    hipArray* cu_ci_array = 0;
    cu_err = hipGraphicsSubResourceGetMappedArray(&cu_ci_array, cmap_image_res, 0, 0);
    cu_err = hipBindTextureToArray(colormap_texture, cu_ci_array);

    // calculate the grid and block sizes
    //hipFuncAttributes  cu_krnl_attr;
    //cu_err = hipFuncGetAttributes(&cu_krnl_attr, reinterpret_cast<const void*>("main_vrc"));

    dim3 vsize = dim3(out_image_w, out_image_h, 1);
    //dim3 bsize = dim3(32, cu_krnl_attr.maxThreadsPerBlock / 32, 1);
    dim3 bsize = dim3(8, 24, 1);
    dim3 gsize;

    gsize.x = vsize.x % bsize.x == 0 ? vsize.x / bsize.x : (vsize.x / bsize.x + 1);
    gsize.y = vsize.y % bsize.y == 0 ? vsize.x / bsize.x : (vsize.y / bsize.y + 1);

    dim3 grid_size(gsize.x, gsize.y, 1);
    dim3 block_size(bsize.x, bsize.y, 1);

    main_vrc<<<grid_size, block_size, 0, hip_stream>>>(out_image_w, out_image_h, use_supersampling);//, uniform_data_raw);
}

extern "C"
bool
upload_uniform_data(const volume_uniform_data& vud,
                    hipStream_t               hip_stream)
{
    hipError_t cu_err = hipMemcpyToSymbolAsync(HIP_SYMBOL(uniform_data), &vud, sizeof(volume_uniform_data), 0, hipMemcpyHostToDevice, hip_stream);
    return hipSuccess == cu_err;
}
