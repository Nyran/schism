#include "hip/hip_runtime.h"

// Copyright (c) 2012 Christopher Lux <christopherlux@gmail.com>
// Distributed under the Modified BSD License, see license.txt.

#include "volume_ray_cast.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cutil/cutil_math.h>

#include <renderer/volume_uniform_data.h>

#define SCM_LDATA_CUDA_VIS_PROFILE_CLOCK 0 
#define SCM_LDATA_CUDA_VIS_ITER_COUNT    0
#define SCM_LDATA_CUDA_VIS_DEBUG         0

#define SCM_LDATA_CUDA_VIS_SS_COUNT      4 // supported modes: 1, 4, 8

namespace scm {
namespace cuda {

// cuda globals
surface<void, cudaSurfaceType2D> out_image;

texture<unsigned char, hipTextureType3D, hipReadModeNormalizedFloat> volume_texture;
texture<uchar4,        hipTextureType1D, hipReadModeNormalizedFloat> colormap_texture;
//texture<float,  hipTextureType3D, hipReadModeElementType> volume_texture;
//texture<float4, hipTextureType1D, hipReadModeElementType> colormap_texture;

__device__ __constant__ volume_uniform_data uniform_data;

// helpers
inline __device__ float min(float a, float b) { return (a < b) ? a : b; }
inline __device__ float max(float a, float b) { return (a > b) ? a : b; }
inline __device__ float4 min(float4 a, float4 b) { return make_float4(min(a.x, b.x), min(a.y, b.y), min(a.z, b.z), min(a.w, b.w)); }
inline __device__ float4 max(float4 a, float4 b) { return make_float4(max(a.x, b.x), max(a.y, b.y), max(a.z, b.z), max(a.w, b.w)); }
inline __device__ float3 min(float3 a, float3 b) { return make_float3(min(a.x, b.x), min(a.y, b.y), min(a.z, b.z)); }
inline __device__ float3 max(float3 a, float3 b) { return make_float3(max(a.x, b.x), max(a.y, b.y), max(a.z, b.z)); }

inline
__device__ __inline__
float4
mul_matrix4(const float4x4 m, const float4 v)
{
    return make_float4(dot(v, m.rows[0]),  //v), //
                       dot(v, m.rows[1]),  //v), //
                       dot(v, m.rows[2]),  //v), //
                       dot(v, m.rows[3])); //v));//
}                                          
                                           
struct ray                                 
{
    float3  origin;
    float3  direction;
    float3  direction_rec;
}; // struct ray

inline
__device__ __inline__
void
make_ray(struct ray*const r,
         const float2 spos,
         const int2   ssize)
{
    float4 spos_nrm = make_float4((spos.x / (float)ssize.x) * 2.0f - 1.0f,
                                  (spos.y / (float)ssize.y) * 2.0f - 1.0f,
                                  -1.0f,
                                   1.0f);
    //float4 spos_os  = mul_matrix4_ptr(&(vdata->_mvp_matrix_inverse), &spos_nrm);
    float4 spos_os  = mul_matrix4(uniform_data._mvp_matrix_inverse, spos_nrm);
    spos_os /= spos_os.w;

    r->origin        = make_float3(uniform_data._os_camera_position);//.xyz;
    r->direction     = normalize(make_float3(spos_os) - r->origin);//vdata->_mvp_matrix_inverse.s012;//spos_os.xyz;//
    r->direction_rec = 1.0f / r->direction;
}

bool
__device__ __inline__
ray_box_intersection(const struct ray*const r,
                     float3   bbmin,
                     float3   bbmax,
                     float*   tmin,
                     float*   tmax)
{
    // compute intersection of ray with all six bbox planes
    float3 tbot = r->direction_rec * (bbmin - r->origin);
    float3 ttop = r->direction_rec * (bbmax - r->origin);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin_a = min(ttop, tbot);
    float3 tmax_a = max(ttop, tbot);

    // find the largest tmin and the smallest tmax
    *tmin = max(max(tmin_a.x, tmin_a.y), max(tmin_a.x, tmin_a.z));
    *tmax = min(min(tmax_a.x, tmax_a.y), min(tmax_a.x, tmax_a.z));

    return ((*tmin > 0.0) && (*tmax > *tmin));
}

inline
__device__ __inline__
float
length_sqr(const float3 a, const float3 b)
{
    float3 d = b - a;
    //return mad(d.x, d.x, mad(d.y, d.y, d.z * d.z));
    return dot(d, d);
}

template<bool ENABLED>
struct thread_vis_clock
{
    clock_t _thread_start;
    clock_t _thread_stop;
    
    __device__ __inline__ bool enabled() const {
        return ENABLED;
    }
    __device__ __inline__ void start() {
        _thread_start = clock();
    }
    __device__ __inline__ void stop() {
        _thread_stop = clock();
    }
    __device__ __inline__ float4 pseudo_colored_elapsed() const {
        return tex1D(colormap_texture, (float)(_thread_stop - _thread_start) / 3000000.0f);
    }
};

template<>
struct thread_vis_clock<false>
{
    __device__ __inline__ bool enabled() const { return false; }
    __device__ __inline__ void start() {}
    __device__ __inline__ void stop()  {}
    __device__ __inline__ float4 pseudo_colored_elapsed() const { return make_float4(0.0f); }
};


void
__global__
main_vrc(unsigned out_image_w, unsigned out_image_h)
{
    thread_vis_clock<false> thread_clock_vis;

    const int2 osize = make_int2(out_image_w, out_image_h);
    const int2 opos  = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                 blockIdx.y * blockDim.y + threadIdx.y);

    if (opos.x < osize.x && opos.y < osize.y) {
        thread_clock_vis.start();

        float4     out_color = make_float4(0.0);

        float tmin = 0.0;
        float tmax = 0.0;
        
        // setup ray 
        struct ray cur_ray;
        const float2 opos_pc = make_float2(0.5f, 0.5f) + make_float2(opos.x, opos.y);
        make_ray(&cur_ray, opos_pc, osize);

        if (ray_box_intersection(&cur_ray, make_float3(0.0), make_float3(uniform_data._volume_extends), &tmin, &tmax)) {
            float3 cam_pos   = make_float3(uniform_data._os_camera_position);
            float3 ray_entry = tmin * cur_ray.direction + cur_ray.origin;
            float3 ray_exit  = tmax * cur_ray.direction + cur_ray.origin;

            float3 ray_increment = cur_ray.direction * uniform_data._sampling_distance.x;
            float3 sampling_pos  = ray_entry + ray_increment; // test, increment just to be sure we are in the volume
            float3 to_tex        = make_float3(uniform_data._scale_obj_to_tex);

            float smpl_sqr_dist  = length_sqr(cam_pos, sampling_pos);
            float exit_sqr_dist  = length_sqr(cam_pos, ray_exit);

            float4 dst = make_float4(0.0f);
            float  opc = uniform_data._sampling_distance.y;

            //out_color = make_float4(ray_exit, 1.0);
            while ((exit_sqr_dist - smpl_sqr_dist) > 0.0f && dst.w < 0.99f) {
                float3 tc_vol = sampling_pos * to_tex;

                float  s   = tex3D(volume_texture, tc_vol.x, tc_vol.y, tc_vol.z);// texture(volume_raw, sampling_pos * volume_data.scale_obj_to_tex.xyz).r;
                float4 src = tex1D(colormap_texture, s);
                //float4 src    = read_imagef(volume_image, vol_smpl, tc_vol).xxxx;//(float4)(s);//texture(color_map, s);

                //float4 src = (float4)(s, s, s, 0.1);

                // increment ray
                sampling_pos  += ray_increment;
                smpl_sqr_dist  = length_sqr(cam_pos, sampling_pos);

                //float3 d = cam_pos - sampling_pos;
                //smpl_sqr_dist  = dot(d, d);

                //inside_volume  = inside_volume_bounds(sampling_pos) && (dst.a < 0.99);

                // opacity correction
                src.w = 1.0f - pow(1.0f - src.w, opc);

                // compositing
                float omda_sa = (1.0 - dst.w) * src.w;
                dst.x += omda_sa * src.x;
                dst.y += omda_sa * src.y;
                dst.z += omda_sa * src.z;
                dst.w   += omda_sa;
            }
            out_color = dst;
        }


        thread_clock_vis.stop();
        if (thread_clock_vis.enabled()) {
            out_color = thread_clock_vis.pseudo_colored_elapsed();
        }

        uchar4 out_col_data;
        out_col_data.x = (unsigned char)(out_color.x * 255.0f);
        out_col_data.y = (unsigned char)(out_color.y * 255.0f);
        out_col_data.z = (unsigned char)(out_color.z * 255.0f);
        out_col_data.w = (unsigned char)(out_color.w * 255.0f);

        surf2Dwrite(out_col_data, out_image, opos.x * sizeof(uchar4), opos.y);
    }
}

void
startup_ray_cast_kernel(unsigned out_image_w, unsigned out_image_h,
                        hipGraphicsResource_t                   output_image_res,
                        hipGraphicsResource_t                   volume_image_res,
                        hipGraphicsResource_t                   cmap_image_res,
                        hipStream_t                             hip_stream)
{
    hipError_t   cu_err = hipSuccess;

    // output image
    hipArray*             cu_oi_array = 0;
    cu_err = hipGraphicsSubResourceGetMappedArray(&cu_oi_array, output_image_res, 0, 0);
    cu_err = cudaBindSurfaceToArray(out_image, cu_oi_array);

    // volume texture
    volume_texture.addressMode[0] = hipAddressModeClamp;
    volume_texture.addressMode[1] = hipAddressModeClamp;
    volume_texture.addressMode[2] = hipAddressModeClamp;
    volume_texture.filterMode     = hipFilterModeLinear;
    volume_texture.normalized     = true;
    hipArray* cu_vi_array = 0;
    cu_err = hipGraphicsSubResourceGetMappedArray(&cu_vi_array, volume_image_res, 0, 0);
    cu_err = hipBindTextureToArray(volume_texture, cu_vi_array);

    // color map texture
    colormap_texture.addressMode[0] = hipAddressModeClamp;
    colormap_texture.filterMode     = hipFilterModeLinear;
    colormap_texture.normalized     = true;
    hipArray* cu_ci_array = 0;
    cu_err = hipGraphicsSubResourceGetMappedArray(&cu_ci_array, cmap_image_res, 0, 0);
    cu_err = hipBindTextureToArray(colormap_texture, cu_ci_array);

    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(main_vrc), hipFuncCachePreferL1);

    // calculate the grid and block sizes
    //hipFuncAttributes  cu_krnl_attr;
    //cu_err = hipFuncGetAttributes(&cu_krnl_attr, reinterpret_cast<const void*>(main_vrc));

    //std::cout << cu_krnl_attr.maxThreadsPerBlock;

    dim3 vsize = dim3(out_image_w, out_image_h, 1);
    //dim3 bsize = dim3(32, cu_krnl_attr.maxThreadsPerBlock / 32, 1);
    dim3 bsize = dim3(8, 24, 1);
    dim3 gsize;

    gsize.x = vsize.x % bsize.x == 0 ? vsize.x / bsize.x : (vsize.x / bsize.x + 1);
    gsize.y = vsize.y % bsize.y == 0 ? vsize.x / bsize.x : (vsize.y / bsize.y + 1);

    dim3 grid_size(gsize.x, gsize.y, 1);
    dim3 block_size(bsize.x, bsize.y, 1);

    main_vrc<<<grid_size, block_size, 0, hip_stream>>>(out_image_w, out_image_h);
}

bool
upload_uniform_data(const volume_uniform_data& vud,
                    hipStream_t               hip_stream)
{
    hipError_t cu_err = hipMemcpyToSymbolAsync(HIP_SYMBOL(uniform_data), &vud, sizeof(volume_uniform_data), 0, hipMemcpyHostToDevice, hip_stream);
    return hipSuccess == cu_err;
}

} // namespace cuda
} // namespace scm
